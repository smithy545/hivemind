#include <hip/hip_runtime.h>


__device__ float *loadRect(const float *v, int offset) {
    auto rectArr = new float[8];
    rectArr[0] = v[offset];
    rectArr[1] = v[offset];
    rectArr[2] = v[offset+1];
    rectArr[3] = v[offset];
    rectArr[4] = v[offset];
    rectArr[5] = v[offset+1];
    rectArr[6] = v[offset+1];
    rectArr[7] = v[offset+1];
    return rectArr;
}

__device__ void rectCollision(float *v, int numComparisons) {
    for(int i = 0; i < numComparisons; i++) {
        //auto rect1 = loadRect(x, y, 8*i);
        //auto rect2 = loadRect(x, y, 8*i);
        // collision
    }
}
