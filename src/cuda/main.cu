#include <iostream>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int cuda_main() {
    const int N = 10;
    auto in = new float[N];
    auto out = new float[N];

    std::cout << "Before" << std::endl;
    for (int i = 0; i < N; i++) {
        in[i] = i;
        out[i] = 0;
        std::cout << in[i] << " " << out[i] << std::endl;
    }

    hipError_t err = hipSuccess;
    size_t size = N * sizeof(float);

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **) &d_A, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **) &d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **) &d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, in, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, in, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(out, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    vectorAdd<<<1, 1>>>(in, in, out, N);

    std::cout << "After" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << in[i] << " " << out[i] << std::endl;
    }

    delete[] in;
    delete[] out;

    return 0;
}

